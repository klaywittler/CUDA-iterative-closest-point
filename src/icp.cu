#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/reduce.h>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "icp.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "svd3.h"


#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

#define OCTREE 0

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128
/*! Size of the starting area in simulation space. */
#define scene_scale 25.0f

/*****************************
* Self defined configuration *
******************************/
/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
int startSize;
int targetSize;

//dim3 threadsPerBlock(blockSize);
//dim3 startblocksPerGrid(blockSize);
//dim3 targetblocksPerGrid(blockSize);

glm::vec3 *dev_pos;
glm::vec3 *dev_start;
glm::vec3 *dev_target;
glm::vec3 *dev_color;
int *dev_cor;

glm::vec3 *host_start;
glm::vec3 *host_target;
int *cor;

/******************
* initSimulation *
******************/
__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

__global__ void kernColorBuffer(int N, glm::vec3 *intBuffer, glm::vec3 value) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= N) {
		return;
	}
	intBuffer[index] = value;
}

void transformCPU(glm::vec3 *pos, glm::mat3 &R, glm::vec3 &T) {
	for (int i = 0; i < startSize; i++) {
		pos[i] = R * pos[i] + T;
	}
}

__global__ void transform(int n, glm::vec3 *pos, glm::mat3 R, glm::vec3 T) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= n) {
		return;
	}
	pos[index] = R * pos[index] + T;
}


/**
* Initialize memory, update some globals
*/
void ICP::initSimulation(std::vector<glm::vec3> start, std::vector<glm::vec3> target, bool transformScan = true) {
	startSize = start.size();
	targetSize = target.size();
	numObjects = startSize + targetSize;

	dim3 startblocksPerGrid((startSize + blockSize - 1) / blockSize);
	dim3 targetblocksPerGrid((targetSize + blockSize - 1) / blockSize);

	// Don't forget to hipFree in  ICP::endSimulation.
	hipMalloc((void**)&dev_start, startSize * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_start failed!");

	hipMalloc((void**)&dev_target, targetSize * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_target failed!");

	hipMalloc((void**)&dev_pos, numObjects * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

	hipMalloc((void**)&dev_color, numObjects * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_color failed!");

	hipMalloc((void**)&dev_cor, startSize * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

	// move start and target points to GPU
	hipMemcpy(dev_start, &start[0], startSize * sizeof(glm::vec3), hipMemcpyHostToDevice);
	hipMemcpy(dev_target, &target[0], targetSize * sizeof(glm::vec3), hipMemcpyHostToDevice);

	host_start = (glm::vec3*) malloc(startSize * sizeof(glm::vec3));
	host_target = (glm::vec3*) malloc(targetSize * sizeof(glm::vec3));
	cor = (int*) malloc(startSize * sizeof(int));

	memcpy(host_start, &start[0], startSize * sizeof(glm::vec3));
	memcpy(host_target, &target[0], targetSize * sizeof(glm::vec3));

	if (transformScan) {
		//add rotation and translation to start for test;
		glm::vec3 T(5.0, -18.0, 10.0);
		glm::mat3 R = glm::mat3(glm::vec3(0.866, -0.5, 0.0), glm::vec3(0.25, 0.433, -0.866), glm::vec3(0.433, 0.75, 0.5));
		//glm::mat3 R = glm::mat3(glm::vec3(0.866, -0.5, 0.0), glm::vec3(0.5, 0.866, 0), glm::vec3(0.0, 0.0, 1.0));

		// move target set
		transform << <startblocksPerGrid, blockSize >> > (startSize, dev_start, R, T);
		transformCPU(host_start, R, T);
	}

	hipMemcpy(dev_pos, dev_start, startSize * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
	hipMemcpy(&dev_pos[startSize], dev_target, targetSize * sizeof(glm::vec3), hipMemcpyDeviceToDevice);

	//set colors for points
	kernColorBuffer << <startblocksPerGrid, blockSize >> > (startSize, dev_color, glm::vec3(0, 1, 0));
	kernColorBuffer << <targetblocksPerGrid, blockSize >> > (targetSize, &dev_color[startSize], glm::vec3(1, 0, 0));

	hipDeviceSynchronize();
}


void ICP::endSimulation() {
	hipFree(dev_pos);
	hipFree(dev_start);
	hipFree(dev_target);
	hipFree(dev_cor);
	hipFree(dev_color);

	free(host_start);
	free(host_target);
	free(cor);
}


/******************
* copyICPToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyColorToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyICPToVBO CUDA kernel.
*/
void ICP::copyToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyColorToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_color, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyToVBO failed!");

  hipDeviceSynchronize();
}

/******************
* stepSimulation *
******************/
void correspondenceCPU(glm::vec3 *start, glm::vec3 *target) {
	for (int i = 0; i < startSize; i++) {
		float best = glm::distance(start[i], target[0]);
		cor[i] = 0;
		for (int j = 1; j < targetSize; j++) {
			float dist = glm::distance(start[i], target[j]);
			if (dist < best) {
				cor[i] = j;
				best = dist;
			}
		}
	}
}

void outerProductCPU(glm::vec3 *target, glm::vec3 *start, glm::mat3 &product) {
	for (int i = 0; i < startSize; i++) {
		product += glm::outerProduct(target[i], start[i]);
	}
}

void ICP::stepCPU() {
	glm::vec3 *temp_start = (glm::vec3*) malloc(startSize * sizeof(glm::vec3));
	glm::vec3 *temp_target = (glm::vec3*) malloc(targetSize * sizeof(glm::vec3));

	memcpy(temp_start, &host_start[0], startSize * sizeof(glm::vec3));
	memcpy(temp_target, &host_target[0], targetSize * sizeof(glm::vec3));

	glm::vec3 startMu(0.0f);
	glm::vec3 targetMu(0.0f);

	// mean center both data sets
	int i = 0;
	while (i < startSize || i < targetSize) {
		if (i < startSize) {
			startMu += host_start[i];
		}
		if (i < targetSize) {
			targetMu += host_target[i];
		}
		i++;
	}
	startMu /= startSize;
	targetMu /= targetSize;
	i = 0;
	while (i < startSize || i < targetSize) {
		if (i < startSize) {
			temp_start[i] -= startMu;
		}
		if (i < targetSize) {
			temp_target[i] -= targetMu;
		}
		i++;
	}
	   
	// find correspondences
	glm::vec3 *cor_target = (glm::vec3*) malloc(startSize * sizeof(glm::vec3));;
	correspondenceCPU(temp_start, temp_start);
	// shuffle
	for (int i = 0; i < startSize; i++) {
		cor_target[i] = temp_start[cor[i]];
	}
	
	// outer product of cor_target and dev_start for svd
	glm::mat3 M(0.0f), U, S, V;
	outerProductCPU(cor_target, temp_start, M);
	// svd
	svd(M[0][0], M[1][0], M[2][0], M[0][1], M[1][1], M[2][1], M[0][2], M[1][2], M[2][2],
		U[0][0], U[1][0], U[2][0], U[0][1], U[1][1], U[2][1], U[0][2], U[1][2], U[2][2],
		S[0][0], S[1][0], S[2][0], S[0][1], S[1][1], S[2][1], S[0][2], S[1][2], S[2][2],
		V[0][0], V[1][0], V[2][0], V[0][1], V[1][1], V[2][1], V[0][2], V[1][2], V[2][2]
	);

	glm::mat3 I(1.0f);
	I[2][2] = glm::determinant(U*glm::transpose(V));
	// multiply for R, rotation
	glm::mat3 R = U * I * glm::transpose(V);
	glm::vec3 T = targetMu - R * startMu;

	// move start set
	transformCPU(host_start, R, T);
	hipMemcpy(dev_pos, &host_start[0], startSize * sizeof(glm::vec3), hipMemcpyHostToDevice);

	free(temp_start);
	free(temp_target);
	free(cor_target);
}

__global__ void correspondenceGPU(int startSize, int targetSize, glm::vec3 *dev_start, glm::vec3 *dev_target, int *dev_cor){
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= startSize) {
		return;
	}
	float best = glm::distance(dev_start[index], dev_target[0]);
	dev_cor[index] = 0;
	for (int j = 1; j < targetSize; j++) {
		float dist = glm::distance(dev_start[index], dev_target[j]);
		if (dist < best) {
			dev_cor[index] = j;
			best = dist;
		}
	}
}

__global__ void correspondenceOctree(int startSize, int targetSize, glm::vec3 *dev_start, glm::vec3 *dev_target, int *dev_cor) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= startSize) {
		return;
	}
	float best = glm::distance(dev_start[index], dev_target[0]);
	dev_cor[index] = 0;


	for (int j = 1; j < targetSize; j++) {
		float dist = glm::distance(dev_start[index], dev_target[j]);
		if (dist < best) {
			dev_cor[index] = j;
			best = dist;
		}
	}
}

__global__ void shuffleTarget(int n, glm::vec3 *dev_target, glm::vec3 *cor_target, int *cor) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= n) {
		return;
	}
	cor_target[index] = dev_target[cor[index]];
}

__global__ void outerProduct(int n, glm::vec3 *dev_target, glm::vec3 *dev_start, glm::mat3 *product) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= n) {
		return;
	}
	 product[index] = glm::outerProduct(dev_target[index], dev_start[index]);
}

void ICP::stepGPU() {
	dim3 startblocksPerGrid((startSize + blockSize - 1) / blockSize);
	dim3 targetblocksPerGrid((targetSize + blockSize - 1) / blockSize);

	// mean center both data sets
	thrust::device_ptr<glm::vec3> thrust_target(dev_target);
	thrust::device_ptr<glm::vec3> thrust_start(dev_start);

	glm::vec3 targetMu = thrust::reduce(thrust_target, thrust_target + targetSize, glm::vec3(0.0f)) / float(targetSize);
	glm::vec3 startMu = thrust::reduce(thrust_start, thrust_start + startSize, glm::vec3(0.0f)) / float(startSize);

	transform << <targetblocksPerGrid, blockSize >> > (targetSize, dev_target, glm::mat3(1.0f), -targetMu);
	transform << <startblocksPerGrid, blockSize >> > (startSize, dev_start, glm::mat3(1.0f), -startMu);
	checkCUDAErrorWithLine("mean center failed!");

	// find correspondences
	glm::vec3 *cor_target;
	hipMalloc((void**)&cor_target, startSize * sizeof(glm::vec3));

#if OCTREE

#else
	correspondenceGPU <<<startblocksPerGrid, blockSize >>> (startSize, targetSize, dev_start, dev_target, dev_cor);
	checkCUDAErrorWithLine("correspondences failed!");
#endif

	shuffleTarget <<<startblocksPerGrid, blockSize >>> (startSize, dev_target, cor_target, dev_cor);
	checkCUDAErrorWithLine("shuffle failed!");

	// outer product of cor_target and dev_start for svd
	glm::mat3 *dev_M, U, S, V;
	hipMalloc((void**)&dev_M, startSize * sizeof(glm::mat3));
	hipMemset(dev_M, 0.0f, startSize * sizeof(glm::mat3));

	outerProduct << <startblocksPerGrid, blockSize >> > (startSize, cor_target, dev_start, dev_M);
	checkCUDAErrorWithLine("outer product  failed!");

	thrust::device_ptr<glm::mat3> thrust_M(dev_M);
	glm::mat3 M = thrust::reduce(thrust_M, thrust_M + startSize, glm::mat3(0.0f));

	// svd
	svd(M[0][0], M[1][0], M[2][0], M[0][1], M[1][1], M[2][1], M[0][2], M[1][2], M[2][2],
		U[0][0], U[1][0], U[2][0], U[0][1], U[1][1], U[2][1], U[0][2], U[1][2], U[2][2],
		S[0][0], S[1][0], S[2][0], S[0][1], S[1][1], S[2][1], S[0][2], S[1][2], S[2][2],
		V[0][0], V[1][0], V[2][0], V[0][1], V[1][1], V[2][1], V[0][2], V[1][2], V[2][2]
	);

	glm::mat3 I(1.0f);
	I[2][2] = glm::determinant(U*glm::transpose(V));
	// multiply for R, rotation
	glm::mat3 R = U * I * glm::transpose(V);
	glm::vec3 T = targetMu - R * startMu;

	// move start set
	transform << <startblocksPerGrid, blockSize >> > (startSize, dev_pos, R, T);
	hipMemcpy(dev_start, dev_pos, startSize * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
	hipMemcpy(dev_target, &dev_pos[startSize], targetSize * sizeof(glm::vec3), hipMemcpyDeviceToDevice);

	hipFree(cor_target);
	hipFree(dev_M);
	checkCUDAErrorWithLine("free memeory failed!");
}

void ICP::unitTest() {
	//Eigen::MatrixXd m(2, 2);
	//m(0, 0) = 3;
	//m(1, 0) = 2.5;
	//m(0, 1) = -1;
	//m(1, 1) = m(1, 0) + m(0, 1);
	//std::cout << m << std::endl;
	return;
}
