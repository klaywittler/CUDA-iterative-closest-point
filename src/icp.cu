#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "icp.h"
#include <hipblas.h>
//#include <Eigen/Dense>



#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)
#define index(i,j,ld) (((j)*(ld))+(i))

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

/*! Size of the starting area in simulation space. */
#define scene_scale 25.0f

/*****************************
* Self defined configuration *
******************************/
/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
int startSize;
int targetSize;

dim3 threadsPerBlock(blockSize);

glm::vec3 *dev_pos;
glm::vec3 *dev_color;
int *dev_cor;

glm::vec3 *pos;
int *cor;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

__global__ void kernColorBuffer(int N, glm::vec3 *intBuffer, glm::vec3 value) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N) {
		intBuffer[index] = value;
	}
}
/**
* Initialize memory, update some globals
*/
void ICP::initSimulation(std::vector<glm::vec3> start, std::vector<glm::vec3> target) {
	startSize = start.size();
	targetSize = target.size();
	numObjects = startSize + targetSize;
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

	// LOOK-1.2 - This is basic CUDA memory management and error checking.
	// Don't forget to hipFree in  ICP::endSimulation.
	hipMalloc((void**)&dev_pos, numObjects * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

	hipMalloc((void**)&dev_color, numObjects * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_color failed!");

	hipMalloc((void**)&dev_cor, numObjects * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

	// move start and target points to GPU
	hipMemcpy(dev_pos, &start[0], startSize * sizeof(glm::vec3), hipMemcpyHostToDevice);
	hipMemcpy(&dev_pos[startSize], &target[0], targetSize * sizeof(glm::vec3), hipMemcpyHostToDevice);

	//set colors for points
	dim3 startBlocks((numObjects + blockSize - 1) / blockSize);
	dim3 targetBlocks((numObjects + blockSize - 1) / blockSize);
	kernColorBuffer << <startBlocks, blockSize >> > (startSize, dev_color, glm::vec3(0, 1, 0));
	kernColorBuffer << <targetBlocks, blockSize >> > (targetSize, &dev_color[startSize], glm::vec3(1, 0, 0));

	hipDeviceSynchronize();

	pos = (glm::vec3*)malloc(numObjects * sizeof(glm::vec3));
	cor = (int*)malloc(numObjects * sizeof(int));
}


/******************
* copyICPToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyColorToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyICPToVBO CUDA kernel.
*/
void ICP::copyToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyColorToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_color, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

typedef struct _matrixSize {
	int WA, HA, WB, HB, WC, HC;
} sMatrixSize;

void matrixMultiply(hipblasHandle_t* handle, sMatrixSize &matrix_size, float *d_A, float *d_B, float *d_C) {
	const float alpha = 1.0f;
	const float beta = 0.0f;
	hipblasSgemm(*handle, HIPBLAS_OP_N, HIPBLAS_OP_N, matrix_size.WA, matrix_size.WB, matrix_size.HA, &alpha, d_A, matrix_size.HA, d_B, matrix_size.HB, &beta, d_C, matrix_size.HC);
	checkCUDAError("matrix multiply");
}

void printMat(float*P, int uWP, int uHP) {
	int i, j;
	for (i = 0; i < uHP; i++) {
		for (j = 0; j < uWP; j++)
			printf(" %f ", P[index(i, j, uHP)]);
		printf("\n");
	}
}

void correspondenceCPU() {
	for (int i = 0; i < startSize; i++) {
		float best = glm::distance(pos[i], pos[startSize+i]);
		cor[i] = 0; //startSize
		for (int j = 1; j < targetSize; j++) {
			float dist = glm::distance(pos[i], pos[startSize + j]);
			if (dist < best) {
				cor[i] = j; // j + startSize
				best = dist;
			}

		}
	}
}

void procrustesCPU() {

}

void ICP::stepCPU() {
	glm::vec3 mu_start(0.0f);
	glm::vec3 mu_target(0.0f);

	correspondenceCPU();
	procrustesCPU();

}


__global__ void correspondenceGPU(int n, int targetSize, glm::vec3 *pos, int *cor){
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= n) {
		return;
	}
	float best = glm::distance(pos[index], pos[n + index]);
	cor[index] = 0; //startSize
	for (int j = 1; j < targetSize; j++) {
		float dist = glm::distance(pos[index], pos[n + j]);
		if (dist < best) {
			cor[index] = j; // j + startSize
			best = dist;
		}

	}
}


void ICP::stepGPU() {
	dim3 fullblocksPerGrid((startSize + blockSize - 1) / blockSize);
	correspondenceGPU << <fullblocksPerGrid, blockSize >> > (startSize, targetSize, dev_pos, dev_cor);
}


void ICP::endSimulation() {
  hipFree(dev_pos);
  hipFree(dev_cor);
  hipFree(dev_color);

  free(pos);
  free(cor);

}


void indexInit(float *data, int size) {
	for (int i = 0; i < size; ++i)
		data[i] = (float)i;
}


void ICP::unitTest() {
	//Eigen::MatrixXd m(2, 2);
	//m(0, 0) = 3;
	//m(1, 0) = 2.5;
	//m(0, 1) = -1;
	//m(1, 1) = m(1, 0) + m(0, 1);
	//std::cout << m << std::endl;

	int HA = 3, WA = 3, HB = 3, WB = 1;
	sMatrixSize matrix_size = { WA, HA, WB, HB, WB, HA };

	// allocate host memory for matrices A and B
	unsigned int size_A = matrix_size.WA * matrix_size.HA;
	unsigned int mem_size_A = sizeof(float) * size_A;
	float *h_A = (float *)malloc(mem_size_A);
	unsigned int size_B = matrix_size.WB * matrix_size.HB;
	unsigned int mem_size_B = sizeof(float) * size_B;
	float *h_B = (float *)malloc(mem_size_B);

	// initialize host memory
	indexInit(h_A, size_A);
	indexInit(h_B, size_B);

	// allocate device memory
	float *d_A, *d_B, *d_C;
	unsigned int size_C = matrix_size.WC * matrix_size.HC;
	unsigned int mem_size_C = sizeof(float) * size_C;

	// allocate host memory for the result
	float *h_C = (float *)malloc(mem_size_C);

	hipMalloc((void **)&d_A, mem_size_A);
	hipMalloc((void **)&d_B, mem_size_B);
	hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_C, mem_size_C);

	// setup execution parameters
	dim3 threads(blockSize, blockSize);
	dim3 grid(matrix_size.HB / threads.x, matrix_size.WA / threads.y);

	// create and start timer
	printf("Computing result using CUBLAS... \n");

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	matrixMultiply(&handle, matrix_size, d_A, d_B, d_C);

	// copy result from device to host
	hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

	// Destroy the handle
	hipblasDestroy(handle);

	printf("\n Matriz A: \n");
	printMat(h_A, matrix_size.WA, matrix_size.HA);
	printf("\n Matriz B: \n ");
	printMat(h_B, matrix_size.WB, matrix_size.HB);
	printf("\n Matriz C: \n");
	printMat(h_C, matrix_size.WC, matrix_size.HC);
	printf("\n\n");

	// clean up memory
	free(h_A);
	free(h_B);
	free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	return;
}
